#include "hip/hip_runtime.h"
#define EIGEN_USE_GPU
#include "cuda_helper.h"
#include "Eigen/Core"

namespace warp2d_kernel_internal
{
#define CLAMP 1
#define VALUE 2
  template <class T, bool NORMALIZED, int BORDER_MODE>
  __global__ void warp2d_kernel(
      T* out, const T* in, const T* displacements, const T border_value,
      int x_size, int y_size, int z_size, int w_size)
  {
    int w = blockIdx.z * blockDim.z + threadIdx.z;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    if( x >= x_size || y >= y_size || w >= w_size )
      return;

    typedef Eigen::Matrix<T,2,1> Vec2;
    typedef Eigen::Matrix<int,2,1> Vec2i;
    typedef Eigen::Matrix<T,4,1> Vec4;
    const int xy_size = x_size*y_size;
    const int xyz_size = xy_size*z_size;
#define IN(w,z,y,x) in[(w)*xyz_size+(z)*xy_size+(y)*x_size+(x)]
#define OUT(w,z,y,x) out[(w)*xyz_size+(z)*xy_size+(y)*x_size+(x)]
#define VECTOR(w,z,y,x) displacements[(w)*2*xy_size+(z)*xy_size+(y)*x_size+(x)]
    Vec2 p1(x,y);
    Vec2 v(VECTOR(w,0,y,x), VECTOR(w,1,y,x));
    if( NORMALIZED )
    {
      v.x() *= x_size;
      v.y() *= y_size;
    }
    Vec2 p2 = p1+v;
    Vec2i p2i = p2.template cast<int>();
    
    T a = p2.x()-p2i.x();
    T b = p2.y()-p2i.y();
    Vec4 weights( (1-a)*(1-b), a*(1-b), (1-a)*b, a*b );
    Vec4 values;

    if( BORDER_MODE == CLAMP )
    {
      int x0, y0, x1, y1, x2, y2, x3, y3;
      x0 = min(x_size-1,max(0,p2i.x()));
      y0 = min(y_size-1,max(0,p2i.y()));
      x1 = min(x_size-1,max(0,p2i.x()+1));
      y1 = min(y_size-1,max(0,p2i.y()));
      x2 = min(x_size-1,max(0,p2i.x()));
      y2 = min(y_size-1,max(0,p2i.y()+1));
      x3 = min(x_size-1,max(0,p2i.x()+1));
      y3 = min(y_size-1,max(0,p2i.y()+1));
      for( int z = 0; z < z_size; ++z )
      {
        values(0) = IN(w,z,y0,x0);
        values(1) = IN(w,z,y1,x1);
        values(2) = IN(w,z,y2,x2);
        values(3) = IN(w,z,y3,x3);
        OUT(w,z,y,x) = values.dot(weights);
      }
    }
    else
    {
      int x0, y0, x1, y1, x2, y2, x3, y3;
      x0 = p2i.x();
      y0 = p2i.y();
      x1 = p2i.x()+1;
      y1 = p2i.y();
      x2 = p2i.x();
      y2 = p2i.y()+1;
      x3 = p2i.x()+1;
      y3 = p2i.y()+1;
      for( int z = 0; z < z_size; ++z )
      {
        if( x0 >= 0 && x3 < x_size && y0 >= 0 && y3 < y_size )
        {
          values(0) = IN(w,z,y0,x0);
          values(1) = IN(w,z,y1,x1);
          values(2) = IN(w,z,y2,x2);
          values(3) = IN(w,z,y3,x3);
          OUT(w,z,y,x) = values.dot(weights);
        }
        else
        {
          OUT(w,z,y,x) = border_value;
        }
      }
    }
#undef IN
#undef OUT
#undef VECTOR
  }

} 
using namespace warp2d_kernel_internal;



template <class T>
void warp2d_gpu( 
    const hipStream_t& stream,
    T* out, const T* in, const T* displacements,
    const T border_value, const int border_mode, bool normalized,
    int x_size, int y_size, int z_size, int w_size)
{

  dim3 block(32,4,1);
  dim3 grid;
  grid.x = divup(x_size,block.x);
  grid.y = divup(y_size,block.y);
  grid.z = divup(w_size,block.z);

  if( normalized )
  {
    if( border_mode == CLAMP )
      warp2d_kernel<T,true,CLAMP><<<grid,block,0,stream>>>(
          out, in, displacements, border_value,
          x_size, y_size, z_size, w_size);
    else
      warp2d_kernel<T,true,VALUE><<<grid,block,0,stream>>>(
          out, in, displacements, border_value,
          x_size, y_size, z_size, w_size);
  }
  else
  {
    if( border_mode == CLAMP )
      warp2d_kernel<T,false,CLAMP><<<grid,block,0,stream>>>(
          out, in, displacements, border_value,
          x_size, y_size, z_size, w_size);
    else
      warp2d_kernel<T,false,VALUE><<<grid,block,0,stream>>>(
          out, in, displacements, border_value,
          x_size, y_size, z_size, w_size);
  }

  CHECK_CUDA_ERROR;

}
template void warp2d_gpu<float>(
    const hipStream_t& stream,
    float* out, const float* in, const float* displacements,
    const float border_value, const int border_mode, bool normalized,
    int x_size, int y_size, int z_size, int w_size);
template void warp2d_gpu<double>(
    const hipStream_t& stream,
    double* out, const double* in, const double* displacements,
    const double border_value, const int border_mode, bool normalized,
    int x_size, int y_size, int z_size, int w_size);

